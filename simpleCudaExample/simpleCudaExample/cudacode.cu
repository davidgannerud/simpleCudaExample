#include "hip/hip_runtime.h"
#include "cudacode.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 16

__global__ void hello()
{
	printf("Hello world! I'm a thread %d in block %d\n", threadIdx.x, blockIdx.x);
}

void runCudaCode(void)
{
	// launch the kernel
	hello <<<NUM_BLOCKS, BLOCK_WIDTH>>>();

	// force the printf()s to flush
	hipDeviceSynchronize();

	printf("That's all!\n");
}
